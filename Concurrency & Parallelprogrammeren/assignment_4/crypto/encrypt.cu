#include "hip/hip_runtime.h"
/* Steven Raaijmakers (10804242) & Marcus van Bergen (10871993) */
/* Program uses caeser chiper to decrypt (and encrypt); parallel & sequential*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include "timer.h"

using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}

// Encryption kernel
__global__ void encryptKernel(int n, char* deviceDataIn, char* deviceDataOut, int *key) {
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    // Check wheter index is in bounds of string length
    if(index < n){
        // use mod to check which keys has to be added
        int getKey = index % (sizeof(key) / sizeof(key[0]));
        deviceDataOut[index] = deviceDataIn[index] + key[getKey];
    }
    return;
}

// Decryption kernel
__global__ void decryptKernel(int n, char* deviceDataIn, char* deviceDataOut, int *key) {
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    // Check wheter index is in bounds of string length
    if(index < n ){
        int getKey = index % (sizeof(key) / sizeof(key[0]));
        deviceDataOut[index] = deviceDataIn[index] - key[getKey];
    }
    return;
}

int fileSize() {
  int size;

  ifstream file ("original.data", ios::in|ios::binary|ios::ate);
  if (file.is_open())
  {
    size = file.tellg();
    file.close();
  }
  else {
    cout << "Unable to open file";
    size = -1;
  }
  return size;
}

int readData(char *fileName, char *data) {

  streampos size;

  ifstream file (fileName, ios::in|ios::binary|ios::ate);
  if (file.is_open())
  {
    size = file.tellg();
    file.seekg (0, ios::beg);
    file.read (data, size);
    file.close();

    cout << "The entire file content is in memory." << endl;
  }
  else cout << "Unable to open file" << endl;
  return 0;
}

int writeData(int size, char *fileName, char *data) {
  ofstream file (fileName, ios::out|ios::binary|ios::trunc);
  if (file.is_open())
  {
    file.write (data, size);
    file.close();

    cout << "The entire file content was written to file." << endl;
    return 0;
  }
  else cout << "Unable to open file";

  return -1;
}

int EncryptSeq (int n, char* data_in, char* data_out, int key[])
{
  int i;
  timer sequentialTime = timer("Sequential encryption");

  int len_array = sizeof(key) / sizeof(key[0]);

  sequentialTime.start();
  // itterate through the entire string and encrypt the number
  for (i=0; i<n; i++) {
      int getKey = i % len_array;
      data_out[i] = data_in[i] + key[getKey];

  }
  sequentialTime.stop();

  cout << fixed << setprecision(6);
  cout << "Encryption (sequential): \t\t" << sequentialTime.getElapsed() << " seconds." << endl;

  return 0;
}

int DecryptSeq (int n, char* data_in, char* data_out, int key[])
{
  int i;
  timer sequentialTime = timer("Sequential decryption");
  //get the length of the keyarrat
  int len_array = sizeof(key) / sizeof(key[0]);

  sequentialTime.start();
  // itterate through the entire string and decrypt the number
  // with the key, found using mod (to know which index should be used)
  for (i=0; i < n; i++) {
      int getKey = i % len_array;
      data_out[i] = data_in[i] - key[getKey];
  }

  sequentialTime.stop();

  cout << fixed << setprecision(6);
  cout << "Decryption (sequential): \t\t" << sequentialTime.getElapsed() << " seconds." << endl;

  return 0;
}


int EncryptCuda (int n, char* data_in, char* data_out, int *key) {
    int len_key = sizeof(key) / sizeof(*key);
    int* c_key = NULL;
    checkCudaCall(hipMalloc((void **) &c_key, len_key * sizeof(int)));
    checkCudaCall(hipMemcpy(c_key, key, len_key*sizeof(int), hipMemcpyHostToDevice));


    int threadBlockSize = 512;

    // allocate the vectors on the GPU
    char* deviceDataIn = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataIn, n * sizeof(char)));
    if (deviceDataIn == NULL) {
        cout << "could not allocate memory!" << endl;
        return -1;
    }
    char* deviceDataOut = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataOut, n * sizeof(char)));
    if (deviceDataOut == NULL) {
        checkCudaCall(hipFree(deviceDataIn));
        cout << "could not allocate memory!" << endl;
        return -1;
    }

    timer kernelTime1 = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    // copy the original vectors to the GPU
    memoryTime.start();
    checkCudaCall(hipMemcpy(deviceDataIn, data_in, n*sizeof(char), hipMemcpyHostToDevice));
    memoryTime.stop();

    // execute kernel
    kernelTime1.start();
    encryptKernel<<<ceil((double)n/threadBlockSize), threadBlockSize>>>(n, deviceDataIn, deviceDataOut, c_key);
    hipDeviceSynchronize();
    kernelTime1.stop();

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    memoryTime.start();
    checkCudaCall(hipMemcpy(data_out, deviceDataOut, n * sizeof(char), hipMemcpyDeviceToHost));
    memoryTime.stop();

    checkCudaCall(hipFree(deviceDataIn));
    checkCudaCall(hipFree(deviceDataOut));

    cout << fixed << setprecision(6);
    cout << "Encrypt (kernel): \t\t" << kernelTime1.getElapsed() << " seconds." << endl;
    cout << "Encrypt (memory): \t\t" << memoryTime.getElapsed() << " seconds." << endl;

   return 0;
}

int DecryptCuda (int n, char* data_in, char* data_out, int *key) {
    int len_key = sizeof(key) / sizeof(*key);
    int* c_key = NULL;
    checkCudaCall(hipMalloc((void **) &c_key, len_key * sizeof(int)));
    checkCudaCall(hipMemcpy(c_key, key, len_key*sizeof(int), hipMemcpyHostToDevice));

    int threadBlockSize = 512;

    // allocate the vectors on the GPU
    char* deviceDataIn = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataIn, n * sizeof(char)));
    if (deviceDataIn == NULL) {
        cout << "could not allocate memory!" << endl;
        return -1;
    }
    char* deviceDataOut = NULL;
    checkCudaCall(hipMalloc((void **) &deviceDataOut, n * sizeof(char)));
    if (deviceDataOut == NULL) {
        checkCudaCall(hipFree(deviceDataIn));
        cout << "could not allocate memory!" << endl;
        return -1;
    }

    timer kernelTime1 = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    // copy the original vectors to the GPU
    memoryTime.start();
    checkCudaCall(hipMemcpy(deviceDataIn, data_in, n*sizeof(char), hipMemcpyHostToDevice));
    memoryTime.stop();

    // execute kernel
    kernelTime1.start();
    decryptKernel<<<ceil((double)n/threadBlockSize), threadBlockSize>>>(n, deviceDataIn, deviceDataOut, c_key);
    hipDeviceSynchronize();
    kernelTime1.stop();

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    memoryTime.start();
    checkCudaCall(hipMemcpy(data_out, deviceDataOut, n * sizeof(char), hipMemcpyDeviceToHost));
    memoryTime.stop();

    checkCudaCall(hipFree(deviceDataIn));
    checkCudaCall(hipFree(deviceDataOut));

    cout << fixed << setprecision(6);
    cout << "Decrypt (kernel): \t\t" << kernelTime1.getElapsed() << " seconds." << endl;
    cout << "Decrypt (memory): \t\t" << memoryTime.getElapsed() << " seconds." << endl;

   return 0;
}

int main(int argc, char* argv[]) {
    int key[7] = {0, 4, 5, 6, 1, 2, 3};
    int n;

    n = fileSize();
    if (n == -1) {
	cout << "File not found! Exiting ... " << endl;
	exit(0);
    }

    char* data_in = new char[n];
    char* data_out = new char[n];
    readData("original.data", data_in);

    cout << "Encrypting a file of " << n << " characters." << endl;

    EncryptSeq(n, data_in, data_out, key);
    writeData(n, "sequential.data", data_out);
    EncryptCuda(n, data_in, data_out, key);
    writeData(n, "cuda.data", data_out);

    readData("cuda.data", data_in);

    cout << "Decrypting a file of " << n << "characters" << endl;
    DecryptSeq(n, data_in, data_out, key);
    writeData(n, "sequential_decrypted.data", data_out);
    DecryptCuda(n, data_in, data_out, key);
    writeData(n, "recovered.data", data_out);

    delete[] data_in;
    delete[] data_out;

    return 0;
}
