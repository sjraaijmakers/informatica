#include "hip/hip_runtime.h"
/* Steven Raaijmakers (10804242) & Marcus van Bergen (10871993) */
/* Program uses cuda to simulate wave equation. */
/* Sources: http://bit.ly/2dGeceE, http://bit.ly/2h0nViZ */
/* http://bit.ly/1hRMNnA */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include "file.h"
#include "timer.h"
#include "simulate.h"
#include <iostream>

using namespace std;

static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}

// Calc function
__global__ void calc(double *old, double *cur, double *next, int MAX){
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;

    double c = 0.15;

    // Borders must be 0
    if((index > 0) && (index < MAX)){
        next[index] = 2.0 * cur[index] - old[index] +
                      c * (cur[index - 1] - (2 * cur[index] - cur[index + 1]));
    }
    else if(index == 0 || index == MAX){
        next[index] = 0.0;
    }
    // If index is not within i_max range
    return;
}

double *simulate(const int i_max, const int t_max, double *old,
                 double *cur, double *next){
    int threadBlockSize = 512;
    // old array
    double* old_c = NULL;
    checkCudaCall(hipMalloc((void **)&old_c, i_max * sizeof(double)));
    if(old_c == NULL){
        cout << "could not allocate memory!" << endl;
        exit(1);
    }

    // current array
    double* cur_c = NULL;
    checkCudaCall(hipMalloc((void **)&cur_c, i_max * sizeof(double)));
    if(cur_c == NULL){
        checkCudaCall(hipFree(old_c));
        cout << "could not allocate memory!" << endl;
        exit(1);
    }

    // next array
    double* next_c = NULL;
    checkCudaCall(hipMalloc((void **)&next_c, i_max * sizeof(double)));
    if(next_c == NULL){
        checkCudaCall(hipFree(old_c));
        checkCudaCall(hipFree(cur_c));
        cout << "could not allocate memory!" << endl;
        exit(1);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // fill ca
    checkCudaCall(hipMemcpy(old_c, old, i_max * sizeof(double),
                             hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(cur_c, cur, i_max * sizeof(double),
                             hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(next_c, next, i_max * sizeof(double),
                             hipMemcpyHostToDevice));

    hipEventRecord(start, 0);

    // T complexity
    for(int t = 0; t < t_max; t++){
        // Call cuda
        calc<<<i_max/threadBlockSize, threadBlockSize>>>(old_c, cur_c, next_c,
                                                         i_max - 1);
        // Error check
        checkCudaCall(hipGetLastError());
        // Swap arrays
        double *tmp = old_c;
        old_c = cur_c;
        cur_c = next_c;
        next_c = tmp;
    }
    hipEventRecord(stop, 0);

    // copy cuda array to "normal" array
    checkCudaCall(hipMemcpy(cur, cur_c, i_max * sizeof(double),
                             hipMemcpyDeviceToHost));

    // free cuda mallocs
    checkCudaCall(hipFree(old_c));
    checkCudaCall(hipFree(cur_c));
    checkCudaCall(hipFree(next_c));

    // time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    cout << "kernel invocation took " << elapsedTime << " milliseconds" << endl;

    return cur;
}


// COPY: (nothing changed yet)

typedef double (*func_t)(double x);

double gauss(double x){
    return exp((-1 * x * x) / 2);
}

void fill(double *array, int offset, int range, double sample_start,
        double sample_end, func_t f)
{
    int i;
    float dx;

    dx = (sample_end - sample_start) / range;
    for (i = 0; i < range; i++) {
        array[i + offset] = f(sample_start + i * dx);
    }
}

// checks if x is power of two
int powerOfTwo(int x){
    return !(x == 0) && !(x & (x - 1));
}

int main(int argc, char *argv[])
{
    double *old, *current, *next;
    int t_max, i_max;

    /* Parse commandline args: i_max t_max */
    if (argc < 3) {
        printf("Usage: %s i_max t_max \n", argv[0]);
        printf(" - i_max: number of discrete amplitude points, should be >2\n");
        printf(" - t_max: number of discrete timesteps, should be >=1\n");
        return EXIT_FAILURE;
    }

    i_max = atoi(argv[1]);
    // i_max should be a power of 2
    if(!powerOfTwo(i_max)){
        printf("argument error: i_max should be 2^X.\n");
        exit(0);
    }
    t_max = atoi(argv[2]);

    if (i_max < 3) {
        printf("argument error: i_max should be >2.\n");
        return EXIT_FAILURE;
    }
    if (t_max < 1) {
        printf("argument error: t_max should be >=1.\n");
        return EXIT_FAILURE;
    }

    /* Allocate and initialize buffers. */
    old = (double*)malloc(i_max * sizeof(double));
    current = (double*)malloc(i_max * sizeof(double));
    next = (double*)malloc(i_max * sizeof(double));

    if (old == NULL || current == NULL || next == NULL) {
        fprintf(stderr, "Could not allocate enough memory, aborting.\n");
        return EXIT_FAILURE;
    }

    memset(old, 0, i_max * sizeof(double));
    memset(current, 0, i_max * sizeof(double));
    memset(next, 0, i_max * sizeof(double));


    /* How should we will our first two generations? */
    if (argc > 4) {
        if (strcmp(argv[4], "sin") == 0) {
            fill(old, 1, i_max/4, 0, 2*3.14, sin);
            fill(current, 2, i_max/4, 0, 2*3.14, sin);
        } else if (strcmp(argv[4], "sinfull") == 0) {
            fill(old, 1, i_max-2, 0, 10*3.14, sin);
            fill(current, 2, i_max-3, 0, 10*3.14, sin);
        } else if (strcmp(argv[4], "gauss") == 0) {
            fill(old, 1, i_max/4, -3, 3, gauss);
            fill(current, 2, i_max/4, -3, 3, gauss);
        } else if (strcmp(argv[4], "file") == 0) {
            if (argc < 7) {
                printf("No files specified!\n");
                return EXIT_FAILURE;
            }
            file_read_double_array(argv[5], old, i_max);
            file_read_double_array(argv[6], current, i_max);
        } else {
            printf("Unknown initial mode: %s.\n", argv[4]);
            return EXIT_FAILURE;
        }
    } else {
        /* Default to sinus. */
        fill(old, 1, i_max/4, 0, 2*3.14, sin);
        fill(current, 2, i_max/4, 0, 2*3.14, sin);
    }

    // double time;
    // timer_start();

    timer vectorAddTimer("vector add timer");

    vectorAddTimer.start();

    simulate(i_max, t_max, old, current, next);

    vectorAddTimer.stop();

    cout << vectorAddTimer;

    file_write_double_array("result.txt", current, i_max);

    free(old);
    free(current);
    free(next);

    return EXIT_SUCCESS;
}
