#include "hip/hip_runtime.h"
/*
 * assign1_1.c
 *
 * Contains code for setting up and finishing the simulation.
 * NOTE: YOU SHOULD IMPLEMENT NOT HAVE TO LOOK HERE, IMPLEMENT YOUR CODE IN
 *       simulate.c.
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include "file.h"
#include "timer.h"
#include "simulate.h"
#include <iostream>

double c = 0.15;

using namespace std;

static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}

__global__ void calc(int step, int max, double *old, double *cur, double *next){
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;

    int i_min = step * index;
    int i_max = ((index + 1) * step) - 1;

    // TODO: laatste moet I_MAX == i_max hebben

    for(int i = i_min; i < i_max; i++){
        next[i] = 2.0 * current[i] - old[i] + c * (cur[i - 1] - (2.0 * cur[i] - cur[i + 1]));
    }
}

double *simulate(const int i_max, const int t_max, const int num_threads,
        double *old, double *cur, double *next){

    double *old_c, *cur_c, *next_c;

    // allocate
    checkCudaCall(hipMalloc(&old_c, i_max * sizeof(double)));
    checkCudaCall(hipMalloc(&cur_c, i_max * sizeof(double)));
    checkCudaCall(hipMalloc(&new_c, i_max * sizeof(double)));

    // fill
    // copy data to the vectors
    checkCudaCall(hipMemcpy(old_c, old, i_max * sizeof(double), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(cur_c, cur, i_max * sizeof(double), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(next, next, i_max * sizeof(double), hipMemcpyHostToDevice));

    int step = i_max / num_threads;

    for(int t = 0; t < t_max; t++){
        calc<<<step, num_threads>>>(step, i_max - 1, old, cur, next);

        double *tmp = old;
        old = cur;
        cur = next;
        next = tmp;
    }

    // copy cuda array to "normal" array
    checkCudaCall(hipMemcpy(cur, cur_c, i_max * sizeof(double), hipMemcpyDeviceToHost));

    // free cuda mallocs
    checkCudaCall(hipFree(old));
    checkCudaCall(hipFree(cur));
    checkCudaCall(hipFree(next));

    return current_array;
}


// ASSIGN COPY: (nothing changed yet) 

typedef double (*func_t)(double x);

double gauss(double x){
    return exp((-1 * x * x) / 2);
}

void fill(double *array, int offset, int range, double sample_start,
        double sample_end, func_t f)
{
    int i;
    float dx;

    dx = (sample_end - sample_start) / range;
    for (i = 0; i < range; i++) {
        array[i + offset] = f(sample_start + i * dx);
    }
}


int main(int argc, char *argv[])
{
    double *old, *current, *next;
    int t_max, i_max, num_threads;
    double time;

    /* Parse commandline args: i_max t_max num_threads */
    if (argc < 4) {
        printf("Usage: %s i_max t_max num_threads [initial_data]\n", argv[0]);
        printf(" - i_max: number of discrete amplitude points, should be >2\n");
        printf(" - t_max: number of discrete timesteps, should be >=1\n");
        printf(" - num_threads: number of threads to use for simulation, "
                "should be >=1\n");
        printf(" - initial_data: select what data should be used for the first "
                "two generation.\n");
        printf("   Available options are:\n");
        printf("    * sin: one period of the sinus function at the start.\n");
        printf("    * sinfull: entire data is filled with the sinus.\n");
        printf("    * gauss: a single gauss-function at the start.\n");
        printf("    * file <2 filenames>: allows you to specify a file with on "
                "each line a float for both generations.\n");

        return EXIT_FAILURE;
    }

    i_max = atoi(argv[1]);
    t_max = atoi(argv[2]);
    num_threads = atoi(argv[3]);

    if (i_max < 3) {
        printf("argument error: i_max should be >2.\n");
        return EXIT_FAILURE;
    }
    if (t_max < 1) {
        printf("argument error: t_max should be >=1.\n");
        return EXIT_FAILURE;
    }
    if (num_threads < 1) {
        printf("argument error: num_threads should be >=1.\n");
        return EXIT_FAILURE;
    }

    /* Allocate and initialize buffers. */
    old = malloc(i_max * sizeof(double));
    current = malloc(i_max * sizeof(double));
    next = malloc(i_max * sizeof(double));

    if (old == NULL || current == NULL || next == NULL) {
        fprintf(stderr, "Could not allocate enough memory, aborting.\n");
        return EXIT_FAILURE;
    }

    memset(old, 0, i_max * sizeof(double));
    memset(current, 0, i_max * sizeof(double));
    memset(next, 0, i_max * sizeof(double));

    /* How should we will our first two generations? */
    if (argc > 4) {
        if (strcmp(argv[4], "sin") == 0) {
            fill(old, 1, i_max/4, 0, 2*3.14, sin);
            fill(current, 2, i_max/4, 0, 2*3.14, sin);
        } else if (strcmp(argv[4], "sinfull") == 0) {
            fill(old, 1, i_max-2, 0, 10*3.14, sin);
            fill(current, 2, i_max-3, 0, 10*3.14, sin);
        } else if (strcmp(argv[4], "gauss") == 0) {
            fill(old, 1, i_max/4, -3, 3, gauss);
            fill(current, 2, i_max/4, -3, 3, gauss);
        } else if (strcmp(argv[4], "file") == 0) {
            if (argc < 7) {
                printf("No files specified!\n");
                return EXIT_FAILURE;
            }
            file_read_double_array(argv[5], old, i_max);
            file_read_double_array(argv[6], current, i_max);
        } else {
            printf("Unknown initial mode: %s.\n", argv[4]);
            return EXIT_FAILURE;
        }
    } else {
        /* Default to sinus. */
        fill(old, 1, i_max/4, 0, 2*3.14, sin);
        fill(current, 2, i_max/4, 0, 2*3.14, sin);
    }

    timer_start();

    /* Call the actual simulation that should be implemented in simulate.c. */
    simulate(i_max, t_max, num_threads, old, current, next);

    time = timer_end();
    printf("Took %g seconds\n", time);
    printf("Normalized: %g seconds\n", time / (i_max * t_max));

    file_write_double_array("result.txt", current, i_max);

    free(old);
    free(current);
    free(next);

    return EXIT_SUCCESS;
}
